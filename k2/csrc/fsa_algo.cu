/**
 * @brief fsa_algo  Implementation of FSA algorithm wrappers from fsa_algo.h

 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include "k2/csrc/fsa_algo.h"
#include "k2/csrc/host_shim.h"

// this contains a subset of the algorithms in fsa_algo.h; currently it just
// contains one that are wrappings of the corresponding algorithms in
// host/.
namespace k2 {



}  // namespace k2
